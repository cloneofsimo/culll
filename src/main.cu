#include <sstream>
#include <iostream>
#include <hip/hip_runtime.h>
#include <pybind11/pybind11.h>
#include <pybind11/numpy.h>
#include <pybind11/stl.h>
// #include <big_cuops.h>
#include <gpu_utils.h>
#include "ops/add.cu"
#include "ops/morph.cu"
#include "ops/mult.cu"
#include <big_tensor.h>

using lint = unsigned int;

namespace py = pybind11;

PYBIND11_MODULE(culll, m)
{
  m.def("badd", batchBigTensorAddWrapper);
  m.def("bmult", batchBigTensorMultWrapper);
  m.def("bnegate", batchBigTensorNegateWrapper);
  m.def("bdigit_resize", batchBigTensorDigitResizeWrapper);

  py::class_<BigTensor>(m, "BigTensor")
    .def(py::init<pybind11::array_t<lint>, lint>())
    .def("copy", &BigTensor::copy)
    .def("print_slice", &BigTensor::print_slice)
    .def("add_gpu", &BigTensor::add_gpu)
    .def("mult_gpu", &BigTensor::mult_gpu)
    .def("negate_gpu", &BigTensor::negate_gpu)
    .def("negate_gpu_inplace", &BigTensor::negate_gpu_inplace)
    .def("get_shift_amount_gpu", &BigTensor::get_shift_amount_gpu)
    .def("shift_gpu_inplace", &BigTensor::shift_gpu_inplace)
    .def("redigit_gpu", &BigTensor::redigit_gpu)
    .def("write_numpy", &BigTensor::write_numpy)
    .def("size", &BigTensor::size)
    .def("at_index", &BigTensor::at_index);
}