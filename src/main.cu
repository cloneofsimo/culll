#include <sstream>
#include <iostream>
#include <hip/hip_runtime.h>
#include <pybind11/pybind11.h>
#include <pybind11/numpy.h>
#include <pybind11/stl.h>
#include "ops/add.cu"
#include "ops/mult.cu"

using lint = unsigned int;



PYBIND11_MODULE(culll, m)
{
  m.def("bignumadd", batchLongTensorAddWrapper);
  m.def("bignummult", batchLongTensorMultWrapper);
}