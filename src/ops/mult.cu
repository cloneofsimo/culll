#include "hip/hip_runtime.h"
#include <gpu_utils.h>
#include <big_cuops.h>

/// @brief Naive batchwise bigint multiplciation kernel.
/// @param batched_data_a B x N x M x n
/// @param batched_data_b B x N x M x n
/// @param output_data B x N x M x n
/// @param B
/// @param N
/// @param M
/// @param n is the number of bits allocated.
/// @param lens is number of bits to actually use. In our
/// case, typically lens <= n / 2.
/// @param base
/// @return
__global__ void batchLongTensorOffsetMult(lint *batched_data_a,
                                          lint *batched_data_b,
                                          lint *output_data, lint B, lint N,
                                          lint M, lint n, lint a_start,
                                          lint b_start, lint out_start,
                                          lint a_n, lint b_n, lint base = 10){

    int batch_idx = blockIdx.x * blockDim.x + threadIdx.x;
    int row_idx = blockIdx.y * blockDim.y + threadIdx.y;
    int col_idx = blockIdx.z * blockDim.z + threadIdx.z;
    // use for loops to iterate over n

    lint sum = 0;
    lint overflow = 0;

    int pos = batch_idx * N * M * n + row_idx * M * n + col_idx * n;

    for (int i = 0; i < a_n; i++) {
        for (int j = 0; j < b_n; j++) {
            lint a = batched_data_a[pos + i + a_start];
            lint b = batched_data_b[pos + j + b_start];
            sum = a * b + overflow;
            if (sum >= base) {
                overflow = sum / base;
                sum %= base;
            } else {
                overflow = 0;
            }
            //
            output_data[pos + i + j + out_start] += sum;
        }
        output_data[pos + i + b_n + out_start] += overflow;
        overflow = 0;
    }
    for(int i = out_start; i < out_start + a_n + b_n; i++){
        if(output_data[pos + i] >= base){
            output_data[pos + i + 1] += output_data[pos + i] / base;
            output_data[pos + i] %= base;
        }
    }
}

void batchLongTensorMultWrapper(pybind11::array_t<lint> batched_data_a,
                                pybind11::array_t<lint> batched_data_b,
                                pybind11::array_t<lint> output_data,
                                int mode = 1, int verbose = 0, int base = 10) {
    pybind11::buffer_info ha = batched_data_a.request();
    pybind11::buffer_info hb = batched_data_b.request();
    pybind11::buffer_info hc = output_data.request();

    if (ha.ndim != 4) {
        std::stringstream strstr;
        strstr << "ha.ndim != 4" << std::endl;
        strstr << "ha.ndim: " << ha.ndim << std::endl;
        throw std::runtime_error(strstr.str());
    }

    if (hb.ndim != 4) {
        std::stringstream strstr;
        strstr << "hb.ndim != 4" << std::endl;
        strstr << "hb.ndim: " << hb.ndim << std::endl;
        throw std::runtime_error(strstr.str());
    }

    if (hc.ndim != 4) {
        std::stringstream strstr;
        strstr << "hc.ndim != 4" << std::endl;
        strstr << "hc.ndim: " << hc.ndim << std::endl;
        throw std::runtime_error(strstr.str());
    }

    if (verbose) {

        if (mode == 0) {
            std::cout << "Using Unoptimized Mode" << std::endl;
        } else {
            std::cout << "Using Optimized Mode" << std::endl;
        }

        std::cout << "ha.shape[0]: " << ha.shape[0] << std::endl;
        std::cout << "ha.shape[1]: " << ha.shape[1] << std::endl;
        std::cout << "ha.shape[2]: " << ha.shape[2] << std::endl;
        std::cout << "ha.shape[3]: " << ha.shape[3] << std::endl;

        std::cout << "hb.shape[0]: " << hb.shape[0] << std::endl;
        std::cout << "hb.shape[1]: " << hb.shape[1] << std::endl;
        std::cout << "hb.shape[2]: " << hb.shape[2] << std::endl;
        std::cout << "hb.shape[3]: " << hb.shape[3] << std::endl;

        std::cout << "ha size: " << ha.size * sizeof(lint) << std::endl;
    }

    // reshape hc

    int B, N, M, n;

    B = hc.shape[0];
    N = hc.shape[1];
    M = hc.shape[2];
    n = hc.shape[3];
    lint n_ = n / 2;

    lint *gpu_ptr_a;
    lint *gpu_ptr_b;
    lint *gpu_ptr_c;

    gpuErrchk(hipMalloc(&gpu_ptr_a, ha.size * sizeof(lint)));
    gpuErrchk(hipMalloc(&gpu_ptr_b, hb.size * sizeof(lint)));
    gpuErrchk(hipMalloc(&gpu_ptr_c, hc.size * sizeof(lint)));

    gpuErrchk(hipMemcpy(gpu_ptr_a, ha.ptr, ha.size * sizeof(lint),
                         hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(gpu_ptr_b, hb.ptr, hb.size * sizeof(lint),
                         hipMemcpyHostToDevice));

    dim3 dimBlock(1, 1, 1);
    dim3 dimGrid(B, N, M);
    if (mode == 0) {
        batchLongTensorOffsetMult<<<dimGrid, dimBlock>>>(
            gpu_ptr_a, gpu_ptr_b, gpu_ptr_c, B, N, M, n, 0, 0, 0, n_, n_, base);
    } else {
        std::cout << "Not implemented yet" << std::endl;
    }

    lint *ptr = reinterpret_cast<lint *>(hc.ptr);
    gpuErrchk(hipMemcpy(ptr, gpu_ptr_c, hc.size * sizeof(lint),
                         hipMemcpyDeviceToHost));

    hipFree(gpu_ptr_a);
    hipFree(gpu_ptr_b);
    hipFree(gpu_ptr_c);
}
