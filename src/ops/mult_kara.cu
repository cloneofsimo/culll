#include "hip/hip_runtime.h"

#include <big_cuops.h>
#include <gpu_utils.h>


using lint = unsigned int;


/// @brief D&C Karatsuba multiplication, all pointers have
/// to be on GPU.
/// @param batch_a
/// @param batch_b
/// @param batch_out
/// @param B
/// @param N
/// @param M
/// @param n
/// @param lens
/// @param base
void batchKaratsuba(lint *batch_x, lint *batch_y, lint *batch_out, lint B,
                    lint N, lint M, lint n, lint a_start, lint b_start,
                    lint out_start, lint a_len, lint b_len, lint base) {

    // where to clear gpu memeory?
    int n = std::max(a_len, b_len);
    lint n_lower = n / 2;
    lint n_upper = n - n_lower;

    if (a_len < 4 || b_len < 4) {
        // naive multiplication.
        dim3 dimBlock(1, 1, 1);
        dim3 dimGrid(B, N, M);

        batchLongTensorOffsetMult<<<dimGrid, dimBlock>>>(
            batch_x, batch_y, batch_out, B, N, M, n, a_start0, b_start,
            out_start, a_len, b_len, base);

        return;
    } else {

        lint *ac, *bd, *ad_plus_bc, *a_plus_b, *c_plus_d;

        // a : upper
        // b : lower
        // c : upper
        // d : lower
        lint n_ = std::max(n_upper, n_lower) + 1;

        gpuErrchk(hipMalloc(&ac, sizeof(lint) * B * N * M * n));
        gpuErrchk(hipMalloc(&bd, sizeof(lint) * B * N * M * n));
        gpuErrchk(hipMalloc(&ad_plus_bc, sizeof(lint) * B * N * M * n_));
        gpuErrchk(hipMalloc(&a_plus_b, sizeof(lint) * B * N * M * n_));
        gpuErrchk(hipMalloc(&c_plus_d, sizeof(lint) * B * N * M * n_));

        dim3 dimBlock(1, 1, 1);
        dim3 dimGrid(B, N, M);

        // ac
        batchKaratsuba(batch_x, batch_y, ac, B, N, M, 2 * n_upper,
                       a_start + n_lower, b_start + n_lower, 0, n_upper,
                       n_upper, base);

        // bd
        batchKaratsuba(batch_x, batch_y, bd, B, N, M, 2 * n_lower, a_start,
                       b_start, 0, n_lower, n_lower, base);

        // a + b
        batchLongTensorOffsetAdd(batch_x, batch_x, a_plus_b, B, N, M, n,
                                 a_start + n_lower, a_start, 0, n_, base);

        // c + d
        batchLongTensorOffsetAdd(batch_y, batch_y, c_plus_d, B, N, M, n,
                                 b_start + n_lower, b_start, 0, n_, base);

        // (a + b) * (c + d)
        batchKaratsuba(a_plus_b, c_plus_d, ad_plus_bc, B, N, M, 2 * n_, 0, 0, 0,
                       n_, n_, base);
    }
}